#include "hip/hip_runtime.h"
#include "MYLNODE1.cuh"
#include "MYLNODE2.cuh"
#include "MYLNODE3.cuh"


__forceinline__ __device__ float my_sigmoid(const float x)
{
    return 1.0f/(1+exp(-x));
}

__global__ void mylnode1_kernel(const float *x, const float *addNum, const float *mutNum, float *output, int n)
{
   int idx = threadIdx.x + blockDim.x*blockIdx.x;
   if(idx < n)
   {
        output[idx] = my_sigmoid(x[idx]);
        int select_id =  idx%85;
        if(select_id < 2)
        {
            output[idx] = output[idx]*2 - 0.5 + addNum[(idx/85)*2+select_id];
            output[idx] = output[idx]*8;
        }
        if(select_id >= 2 && select_id < 4)
        {
            output[idx] = output[idx]*2;
            output[idx] = output[idx]*output[idx];
            output[idx] = output[idx]*mutNum[(idx/(85*80*80))*2+select_id%2];
        }
   }
}

__global__ void mylnode2_kernel(const float *x, const float *addNum, const float *mutNum, float *output, int n)
{
   int idx = threadIdx.x + blockDim.x*blockIdx.x;
   if(idx < n)
   {
        output[idx] = my_sigmoid(x[idx]);
        int select_id =  idx%85;
        if(select_id < 2)
        {
            output[idx] = output[idx]*2 - 0.5 + addNum[(idx/85)*2+select_id];
            output[idx] = output[idx]*16;
        }
        if(select_id >= 2 && select_id < 4)
        {
            output[idx] = output[idx]*2;
            output[idx] = output[idx]*output[idx];
            output[idx] = output[idx]*mutNum[(idx/(85*40*40))*2+select_id%2];
        }
   }
}

__global__ void mylnode3_kernel(const float *x, const float *addNum, const float *mutNum, float *output, int n)
{
   int idx = threadIdx.x + blockDim.x*blockIdx.x;
   if(idx < n)
   {
        output[idx] = my_sigmoid(x[idx]);
        int select_id =  idx%85;
        if(select_id < 2)
        {
            output[idx] = output[idx]*2 - 0.5 + addNum[(idx/85)*2+select_id];
            output[idx] = output[idx]*32;
        }
        if(select_id >= 2 && select_id < 4)
        {
            output[idx] = output[idx]*2;
            output[idx] = output[idx]*output[idx];
            output[idx] = output[idx]*mutNum[(idx/(85*20*20))*2+select_id%2];
        }
   }
}


static __global__ void printer_val(const float *x, int n)
{
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    if(idx == 0)
    {
        for (size_t i = 0; i < n; i++)
        {
            printf("%f ", x[i]);
        }
        
    }
}


void MYLNODE1_inference(const float *x, const float *addNum, const float *mutNum, float *output, int n, hipStream_t stream)
{
    const int nthreads = 512;
    int grid_size = (n+nthreads-1)/nthreads;
    mylnode1_kernel<<<grid_size, nthreads>>>(x, addNum, mutNum, output, n);
}

void MYLNODE2_inference(const float *x, const float *addNum, const float *mutNum, float *output, int n, hipStream_t stream)
{
    const int nthreads = 512;
    int grid_size = (n+nthreads-1)/nthreads;
    mylnode2_kernel<<<grid_size, nthreads>>>(x, addNum, mutNum, output, n);
}

void MYLNODE3_inference(const float *x, const float *addNum, const float *mutNum, float *output, int n, hipStream_t stream)
{
    const int nthreads = 512;
    int grid_size = (n+nthreads-1)/nthreads;
    mylnode3_kernel<<<grid_size, nthreads>>>(x, addNum, mutNum, output, n);
}

void myprint_val(const float *x, int n, hipStream_t stream)
{
    printer_val<<<1, 1, 0, stream>>>(x, n);
}