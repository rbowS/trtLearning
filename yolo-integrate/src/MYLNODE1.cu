#include "MYLNODE1.cuh"
#include "NvInfer.h"

#include <cassert>
#include <cstring>
#include <vector>

using std::vector;
using std::string;
using namespace nvinfer1;

void MYLNODE1_inference(const float *x, const float *addNum, const float *mutNum, float *output, const int batchSize, int n, hipStream_t stream);

void myprint_val(const float *x, int n, hipStream_t stream);

// MYLNODE1 plugin的特定常量
namespace{

    const char *MYLNODE1_PLUGIN_VERSION{"1"};// 采用的名称要对应上onnx-tensorrt-release-8.0/builtin_op_importers.cpp:5094行定义的名称
    const char *MYLNODE1_PLUGIN_NAME{"MYLNODE1"};
}

// 静态类字段的初始化
PluginFieldCollection MYLNODE1PluginCreator::mFC{};// FieldCollection 字段收集

vector<PluginField> MYLNODE1PluginCreator::mPluginAttributes;

// 实际注册时，注册的是创建器，交给tensorRT管理
REGISTER_TENSORRT_PLUGIN(MYLNODE1PluginCreator);

// 用于序列化插件的Helper function
template <typename T>
void writeToBuffer(char *&buffer, const T& val)
{
    *reinterpret_cast<T*>(buffer) = val;
    buffer += sizeof(T);
}

// 用于反序列化插件的Helper function
template <typename T>
T readFromBuffer(const char *&buffer)
{
    T val = *reinterpret_cast<const T*>(buffer);
    buffer += sizeof(T);
    return val;
}


MYLNODE1Plugin::MYLNODE1Plugin(const string name)
:mLayerName(name)
{
    #if DEBUG
        std::cout<<"create MYLNODE1Plugin"<<std::endl;
    #endif
}


const char *MYLNODE1Plugin::getPluginType() const noexcept
{
    return MYLNODE1_PLUGIN_NAME;
}

const char *MYLNODE1Plugin::getPluginVersion() const noexcept
{
    return MYLNODE1_PLUGIN_VERSION;
}

int MYLNODE1Plugin::getNbOutputs() const noexcept
{
    return 1;
}

// 获取该层的输出维度是多少
nvinfer1::DimsExprs MYLNODE1Plugin::getOutputDimensions(
    int32_t outputIndex, const nvinfer1::DimsExprs *inputs, 
    int32_t nbInputs, nvinfer1::IExprBuilder &exprBuilder
) noexcept
{
    // MYLNODE1ping不改变输入尺寸，所以输出尺寸将与输入尺寸相同
    return *inputs;
}

int MYLNODE1Plugin::initialize() noexcept
{
    return 0;
}

// 行性能测试
int MYLNODE1Plugin::enqueue(
    const nvinfer1::PluginTensorDesc *inputDesc, const nvinfer1::PluginTensorDesc *outputDesc,
    const void* const *inputs, void* const *outputs, void *worksapce, hipStream_t stream
) noexcept
{
    void *output = outputs[0];
    size_t volume = 1;
    //std::cout<<"inputDesc->dims.nbDims: "<<inputDesc->dims.nbDims<<std::endl;
    for (int i = 1; i < inputDesc->dims.nbDims; i++)
    {
        //std::cout<<"inputDesc->dims.d[i]: "<<inputDesc->dims.d[i]<<std::endl;
        volume *= inputDesc->dims.d[i];
    }
    mInputVolume = volume;
    //myprint_val(static_cast<const float*>(inputs[0]), 320, stream);
    MYLNODE1_inference(
        static_cast<const float*>(inputs[0]),
        static_cast<const float*>(inputs[1]),
        static_cast<const float*>(inputs[2]),
        static_cast<float*>(output),
        inputDesc->dims.d[0],
        mInputVolume,
        stream
    );

    return 0;
}

size_t MYLNODE1Plugin::getSerializationSize() const noexcept
{
    return 0;
}

// 该层的参数序列化储存为trtmodel文件
void MYLNODE1Plugin::serialize(void *buffer) const noexcept
{
    char *d = static_cast<char*>(buffer);
    const char *a = d;
    
    assert(d == a+getSerializationSize());
}

// 判断该插件所支持的数据格式和类型
bool MYLNODE1Plugin::supportsFormatCombination(int32_t pos, const PluginTensorDesc* inOut, int32_t nbInputs, int32_t nbOutputs) 
noexcept
{
    auto type = inOut[pos].type;
    auto format = inOut[pos].format;
    if(type == DataType::kFLOAT && format == PluginFormat::kLINEAR)
        return true;
    return false;
}

void MYLNODE1Plugin::terminate() noexcept {}


// 配置插件格式:告诉你目前这个层所采用的数据格式和类型
void MYLNODE1Plugin::configurePlugin(
    const DynamicPluginTensorDesc* in, int32_t nbInputs,
    const DynamicPluginTensorDesc* out, int32_t nbOutputs
) noexcept
{
    auto type = in->desc.type;
    auto format = in->desc.format;
    assert(nbOutputs == 1);
    assert(type == DataType::kFLOAT);
    assert(format == PluginFormat::kLINEAR);
}

void MYLNODE1Plugin::destroy() noexcept
{
    // This gets called when the network containing plugin is destroyed
    delete this;
}

// 克隆插件
IPluginV2DynamicExt* MYLNODE1Plugin::clone() const noexcept
{
    #if DEBUG
        printf("===================克隆插件MyLnode1=================\n");
    #endif
    auto plugin = new MYLNODE1Plugin(mLayerName);
    plugin->setPluginNamespace(mNamespace.c_str());
    return plugin;
}

void MYLNODE1Plugin::setPluginNamespace(const char* libNamespace) noexcept
{
    mNamespace = libNamespace;
}

const char* MYLNODE1Plugin::getPluginNamespace() const noexcept
{
    return mNamespace.c_str();
}

// 插件创建器
MYLNODE1PluginCreator::MYLNODE1PluginCreator()
{
    // 收集PluginField的参数
    mFC.nbFields = mPluginAttributes.size();
    mFC.fields = mPluginAttributes.data();
}

const char* MYLNODE1PluginCreator::getPluginName() const noexcept
{
    return MYLNODE1_PLUGIN_NAME;
}

const char* MYLNODE1PluginCreator::getPluginVersion() const noexcept
{
    return MYLNODE1_PLUGIN_VERSION;
}

const PluginFieldCollection* MYLNODE1PluginCreator::getFieldNames() noexcept
{
    return &mFC;
}

// 创建plugin
IPluginV2* MYLNODE1PluginCreator::createPlugin(const char *name, const PluginFieldCollection *fc) noexcept
{
    return new MYLNODE1Plugin(name);
}

// 反序列化插件参数进行创建
IPluginV2* MYLNODE1PluginCreator::deserializePlugin(const char *name, const void *serialData, size_t serialLength) noexcept
{
    // This object will be deleted when the network is destroyed, which will
    // call MYLNODE1Plugin::destroy()
    return new MYLNODE1Plugin(name);
}

void MYLNODE1PluginCreator::setPluginNamespace(const char* libNamespace) noexcept
{
    mNamespace = libNamespace;
}

const char* MYLNODE1PluginCreator::getPluginNamespace() const noexcept
{
    return mNamespace.c_str();
}